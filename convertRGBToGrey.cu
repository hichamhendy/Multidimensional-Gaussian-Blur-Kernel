#include "hip/hip_runtime.h"

#include "convertRGBToGrey.hpp"

/*
 * CUDA Kernel Device code
 *
 */
__global__ void multiDimensionalBlur(float* input3DData, float* output3DData)
{
    dim3 sliceDimensions(3, 3, 3);

    const int width = d_width;
    const int height = d_height;
    const int channels = d_channel;
    const int duration = d_duration;

    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int t   = threadIdx.z + blockIdx.z * blockDim.z;

    // Calculate the linear thread ID for the current channel
    const unsigned int threadId = (col + row * width + t * width * height) * channels;

    // Iterate over each channel
    for (int c = 0; c < channels; ++c)
    {
        if (row < height && col < width && t < duration)
        {
            float input3dDataSlice[27];
            const unsigned int channelIndex = threadId + c;

            // Extract the slice for the current channel
            _3dSlice(input3DData, sliceDimensions, input3dDataSlice, row, col, t, channelIndex);

            // Calculate blurred pixel value for the current channel's slice
            float blurredPixelValue = _3dGaussianBlurPixel(input3dDataSlice);

            // Store the blurred pixel value in the output array
            output3DData[channelIndex] = blurredPixelValue;

            // Perform any processing here, for now just copy the input to output
            // output3DData[channelIndex] = input3DData[channelIndex]; // Experiment succeeded
        }
    }
}


/**
 * @brief This function effectively extracts a 3D slice of data from the input 3D volume (input3dData) and stores it into the provided output slice (input3dDataSlice). 
 * Each thread handles copying a single element of the slice, ensuring parallelism across the 3D data volume.
 *  TODO: 
    * loop through z=[t_0, t_1, t_2]  
    * determine block id 
    * loop through x, y  +/- from current mapped x,y from block and thread index information  
    * determine thread id  
    * set x,y,z based on 0 with +/- from bottom left of t_0 
 * 
 * @param input3dData 
 * @param sliceDimensions 
 * @param input3dDataSlice 
 * @return void 
 */
__device__ void _3dSlice(float *input3dData, dim3 sliceDimensions, float *input3dDataSlice, int row, int column, int time, int threadId) 
{
    // Get the dimensions of the slice
    int sliceWidth = sliceDimensions.x;
    int sliceHeight = sliceDimensions.y;
    int sliceDepth = sliceDimensions.z;

    // Get the dimensions of the input 3D data
    int dataWidth = d_width;
    int dataHeight = d_height;
    int dataDepth = d_duration; 

    for(int k = 0; k <  sliceDepth/2; k++) 
    {
        for(int i = -sliceWidth/2; i<= sliceWidth/2; i++) 
        {
            for(int j = -sliceHeight/2; j <= sliceHeight/2; j++) 
            {
                const unsigned int x = max(0, min(dataWidth  - 1, column + j));
                const unsigned int y = max(0, min(dataHeight - 1, row    + i));
                const unsigned int z = max(0, min(dataDepth  - 1, time   + k));

                const unsigned int sliceIndex = x + y * dataWidth + z * (dataHeight * dataWidth);

                input3dDataSlice[j + i * sliceHeight + k * (sliceHeight * sliceWidth) ] = input3dData[threadId];
            }
        }
    }
}

__device__ float _3dGaussianBlurPixel(float * input3dDataSlice) 
{
    float pixelValueSum = 0.0f;

    //loop through x = [0,1,2] 
    //loop through y = [0,1,2]  
    //loop through z = [0,1,2] 

    // Loop through x, y, z = [0, 1, 2]
    for (int z = 0; z < 3; ++z)
    {
        for (int y = 0; y < 3; ++y)
        {
            for (int x = 0; x < 3; ++x)
            {
                // Calculate the 1D index for accessing the slice data
                int sliceIndex = x + y * 3 + z * (3 * 3);

                // Apply the mask and accumulate the result
                pixelValueSum += input3dDataSlice[sliceIndex] * d_3d_mask[x + y * 3 + z * (3 * 3)];
            }
        }
    }
    return pixelValueSum / d_mask_weight_sum; // Normalize by the sum of mask weights
}

__host__ std::tuple<float *, float *> allocateDeviceMemory(int width, int height, int duration, int channels)
{
    std::cout << "Allocating GPU device memory\n";
    int stream_size = width * height * duration * channels;
    size_t size = stream_size * sizeof(float);

    // Allocate the device input vector inputDeviceVideoData
    float *inputDeviceVideoData = NULL;
    hipError_t err = hipMalloc((void**) &inputDeviceVideoData, size);  // study: hipMalloc((void**)& dIn, vBytes(hIn))
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector inputDeviceVideoData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector inputDeviceVideoData
    float *outputDeviceVideoData = NULL;
    err = hipMalloc((void**) &outputDeviceVideoData, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector outputDeviceVideoData; (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return {inputDeviceVideoData, outputDeviceVideoData};
}


__host__ void copyFromHostToDevice(float* h_input, float* d_input, float* h_mask_3d, int width, int height, int duration, int channels, int ker_dim)
{
    std::cout << "Copying from Host to Device\n";
    int stream_size = width * height * duration * channels;
    size_t size = stream_size * sizeof(float);

    hipError_t err;
    err = hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int mask_weight_sum {0};
    for(int i = 0; i < ((int) pow(ker_dim, 3)); i++)
        mask_weight_sum += h_mask_3d[i];

    //Allocate device constant symbols for width and height and duration
    hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_duration), &duration, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_channel), &channels, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask_weight_sum), &mask_weight_sum, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_3d_mask), h_mask_3d, ((int) pow(ker_dim,3)) * sizeof(float), 0, hipMemcpyHostToDevice); //  h_mask_3d is already a pointer to the data, so it shouldn't be dereferenced again.

    hipDeviceSynchronize();
}

__host__ void executeKernel(float *inputDeviceVideoData, float* outputDeviceVideoData, int threadsPerBlock, int width, int height,int duration, int channels)
{
    //Launch the convert CUDA Kernel
    std::cout << "Executing kernel\n";

    // Calculate the number of blocks needed in each dimension
    int blocks_x = (width * 1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocks_y = (height * 1 + threadsPerBlock - 1) / threadsPerBlock;
    int blocks_z = (duration + threadsPerBlock - 1) / threadsPerBlock;

    // Define the block dimensions
    dim3 blockSize(threadsPerBlock, threadsPerBlock, threadsPerBlock);

    // Define the grid dimensions
    dim3 gridSize(blocks_x, blocks_y, blocks_z);

    printf("Calling the kernel for video of %d x %d x %d x %d\n", width, height, duration, channels);

    // Launch the kernel with the specified grid and block dimensions
    multiDimensionalBlur<<<gridSize, blockSize>>>(inputDeviceVideoData, outputDeviceVideoData);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch convert kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void copyFromDeviceToHost(float *d_output, float* h_output, int width,int height,int duration, int channels)
{
    std::cout << "Copying from Device to Host\n";
    // Copy the device result int array in device memory to the host result int array in host memory.
    int stream_size = height * width * duration * channels;
    size_t size = stream_size * sizeof(float);

    hipError_t err = hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy array d_output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateMemory(float *d_input, float *d_output)
{
    std::cout << "Deallocating GPU device memory\n";
    hipError_t err = hipFree(d_input);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    std::cout << "Cleaning CUDA device\n";
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__host__ void cleanUpHost(float* input_host_video_data, int width, int height, int duration)
{
    printf("Cleaning the host...\n\n");
/*     for (int i = 0; i < width; ++i) 
    {
        for (int j = 0; j < height; ++j) 
        {
            delete[] input_host_video_data[i][j];
        }
        delete[] input_host_video_data[i];
    } */
    delete[] input_host_video_data;
}

__host__ std::tuple<std::string, std::string, int, int> parseCommandLineArguments(int argc, char *argv[])
{
    std::cout << "Parsing CLI arguments\n";
    int threadsPerBlock = 256;
    int kernelDim = 3;
    std::string inputImage = "fast-x-teaser-trailer-2023-144.mp4";
    std::string outputImage = inputImage + "_gaussianBlurred.mp4";

    for (int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if (option.compare("-i") == 0)
        {
            inputImage = value;
            outputImage = inputImage + "_gaussianBlurred.mp4";
        }
        else if (option.compare("-o") == 0)
        {
            outputImage = value;
        }
        else if (option.compare("-t") == 0)
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if (option.compare("-k") == 0)
        {
            kernelDim = atoi(value.c_str());
        }
    }
    std::cout << "inputImage: " << inputImage << " outputImage: " << outputImage << " threadsPerBlock: " << threadsPerBlock 
        << " KernelDim: " << kernelDim << "\n";
    return {inputImage, outputImage, threadsPerBlock, kernelDim};
}


__host__ std::tuple<int, int, int, int, int, float*> readVideoFromFile(std::string inputFile) 
{
    cv::VideoCapture cap(inputFile);

    if (!cap.isOpened()) {
        std::cerr << "Error: Couldn't open the video file.\n";
        exit(EXIT_FAILURE);
    }

    // Check if the input video file contains playable streams
    if (cap.get(cv::CAP_PROP_FRAME_COUNT) == 0) {
        std::cerr << "Error: Input video file contains no playable streams.\n";
        exit(EXIT_FAILURE);
    }
    
    int width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
    int height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
    int channels = 3; // cap.get(cv::CAP_PROP_CHANNEL); // Video input or Channel Number (only for those cameras that support)  Assuming RGB color space
    int duration = cap.get(cv::CAP_PROP_FRAME_COUNT);
    int fps = cap.get(cv::CAP_PROP_FPS);
    
    float *output_host_video_data = (float *)malloc(sizeof(float) * width * height * channels * duration);

    std::cout << "Video width: " << width << " height: " << height << " duration: " << duration  << " fbs: " << fps << " channel: " << channels << std::endl;

    float *inputHostVideoData = (float *)malloc(sizeof(float) * width * height * channels * duration);

    std::cout << "Memory allocation on host is successfully done" << std::endl;

    cv::Mat frame;
    for (int t = 0; t < duration; ++t) {
        cap >> frame;
        if (frame.empty()) {
            std::cerr << "ERROR! blank frame grabbed.\n";
            cleanUpHost(inputHostVideoData, width, height, duration);
            exit(EXIT_FAILURE);
        }
        
        for (int i = 0; i < width; ++i) {
            for (int j = 0; j < height; ++j) {
                for (int c = 0; c < channels; ++c) {
                    if (frame.empty()) {
                        std::cerr << "Error: Video file does not contain enough frames.\n";
                        std::cout << "stopped at " << i << " " << j << " " << t << std::endl;
                        cleanUpHost(inputHostVideoData, width, height, duration);
                        exit(EXIT_FAILURE);
                    }
                    inputHostVideoData[t * width * height * channels + j * width * channels + i * channels + c] = static_cast<float>(frame.at<cv::Vec3b>(j, i)[c]) / 255.0;  //  at (int row, int col)
                    
                    output_host_video_data[t * width * height * channels + j * width * channels + i * channels + c] = inputHostVideoData[t * width * height * channels + j * width * channels + i * channels + c];
                }
            }
        }
    }

    cap.release();

    return {width, height, duration, channels, fps, inputHostVideoData};
}


__host__ void storeVideoData(float* outputHostVideoData, int width, int height, int duration, std::string outputFile) 
{
    std::cout << "Start storing the result" << std::endl;

    cv::VideoWriter videoWriter;
    int channels = 3; // constant assumption

    // Define the codec and create VideoWriter object
    int codec = cv::VideoWriter::fourcc('H', '2', '6', '4'); // H.264 codec
    double fps = 30.0; // You can adjust the frames per second as needed
    cv::Size frameSize(width, height);

    videoWriter.open(outputFile, codec, fps, frameSize, true);

    if (!videoWriter.isOpened()) {
        std::cerr << "Error: Couldn't open the video writer.\n";
        exit(EXIT_FAILURE);
    }

    cv::Mat frame(height, width, CV_8UC3); // CV_8UC3 for 3-channel image (e.g., RGB)

    for (int t = 0; t < duration; ++t) 
    {
        for (int i = 0; i < width; ++i) {
            for (int j = 0; j < height; ++j) {
                // Iterate over each channel
                for (int c = 0; c < channels; ++c) {
                    float pixelValue = outputHostVideoData[t * width * height * channels + (j * width + i) * channels + c];
                    // Set the pixel value in the frame
                    frame.at<cv::Vec3b>(j, i)[c] = static_cast<uchar>(pixelValue * 255.0);
                }
            }
        } 

        if (frame.empty()) {
            std::cerr << "Error: Video file does not contain enough frames.\n";
            exit(EXIT_FAILURE);
        }
        videoWriter.write(frame);

        // Display the resulting frame    
        cv::imshow("Results Frame", frame);
        // Press  ESC on keyboard to  exit
        char c = (char) cv::waitKey(10);
        if( c == 27 ) 
            break;
    }

    std::cout << "Storing ended. Check results!" << std::endl;
    cv::destroyAllWindows();
    videoWriter.release();
}


__host__ void generate3DGaussian(float* kernel, int dim, int radius) 
{
    float stdev = 1.0;
    float pi = 3.14159265358979323846;
    float constant = 10.0 / (2.0 * pi * stdev * stdev);

    
        for (int k = 0; k < dim; ++k) {
            for (int i = 0; i < dim; ++i) {
                for (int j = 0; j < dim; ++j) {
                    float exponent = -((i - radius) * (i - radius) + (j - radius) * (j - radius) + (k - radius) * (k - radius)) / (2 * stdev * stdev);
                    kernel[k * dim * dim + i * dim + j] = constant * exp(exponent);
                }
            }
        } 
}



__host__ bool checkCudaCaps()
{
    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if(deviceCount == 0)
    {
        std::cerr << "No CUDA-capable device found" << std::endl;
        return false;
    }
    std::cout << "CUDA-capable devices detected: " << deviceCount << std::endl;

    printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
            (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
            (runtimeVersion % 100) / 10);

    for(int device= 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " <<  deviceProp.name << std::endl;
        std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl; 
        std::cout << "Max Threads Per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        int maxThreads = deviceProp.maxThreadsPerMultiProcessor * deviceProp.multiProcessorCount;
        int maxBlocks = maxThreads / deviceProp.maxThreadsPerBlock;
        std::cout << "Max Blocks: " << maxBlocks << std::endl;
    }
  // Min spec is SM 1.0 devices
  bool bVal = checkCudaCapabilities(1, 0);
  return bVal;
}


int main(int argc, char *argv[])
{
    printf("%s Starting...\n\n", argv[0]);

    if (checkCudaCaps() == false)
    {
      exit(EXIT_SUCCESS);
    }

    std::tuple<std::string, std::string, int, int> parsedCommandLineArgsTuple = parseCommandLineArguments(argc, argv);
    std::string input_video = std::get<0>(parsedCommandLineArgsTuple);
    std::string output_video = std::get<1>(parsedCommandLineArgsTuple);
    const int threads_per_block = std::get<2>(parsedCommandLineArgsTuple);
    const int conv_k_dim = std::get<3>(parsedCommandLineArgsTuple);
    try 
    {
        printf("Starting Reading...\n\n");
        std::tuple<int, int, int, int, int, float*> readTuple = readVideoFromFile(input_video);
        int width = std::get<0>(readTuple);
        int height = std::get<1>(readTuple);
        int duration = std::get<2>(readTuple);
        int channels = std::get<3>(readTuple);
        int fps = std::get<4>(readTuple);
        float* input_host_video_data = std::get<5>(readTuple);
        // float output_host_video_data[width][height][duration];
        printf("Reading the video done...\n\n");
        if ((height < 2 * conv_k_dim + 1) || (width < 2 * conv_k_dim + 1)) 
        {
            std::cout << "Image is too small to apply kernel effectively." << std::endl;
            exit(EXIT_FAILURE);
	    }
        float mask_3d[conv_k_dim * conv_k_dim * conv_k_dim];
        int k_radius = floor( conv_k_dim/ 2.0);
        generate3DGaussian(mask_3d, conv_k_dim, k_radius);
        std::cout << "The mask elments are ";
        for(int i = 0; i < ((int) pow(conv_k_dim, 3)); i++)
            std::cout << mask_3d[i] << " ";
        std::cout << std::endl;


        float *output_host_video_data = (float *)malloc(sizeof(float) * width * height * duration * channels);

        std::tuple<float *, float *> memoryTuple = allocateDeviceMemory(width, height, duration, channels);
        float *input_device_video_data = std::get<0>(memoryTuple);
        float *output_device_video_data = std::get<1>(memoryTuple);

        copyFromHostToDevice(input_host_video_data, input_device_video_data, mask_3d, width, height, duration, channels,conv_k_dim);

        executeKernel(input_device_video_data, output_device_video_data, threads_per_block, width, height, duration, channels);

        copyFromDeviceToHost(output_device_video_data, output_host_video_data, width, height, duration, channels);
        deallocateMemory(input_device_video_data, output_device_video_data);

        storeVideoData(output_host_video_data , width, height, duration,  output_video);
        
        cleanUpDevice();
        cleanUpHost(input_host_video_data, width, height, duration);
        delete output_host_video_data;
    }
    catch (std::exception &error_)
    {
        std::cout << "Caught exception: " << error_.what() << std::endl;
        return 1;
    }
    return 0;
}